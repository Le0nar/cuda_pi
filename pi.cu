#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <omp.h>

// Количество прямоугольников для численного интегрирования
#define KS 100000

// Объявляем количество нитей и блоков
#define THREADS 10
#define BLOCKS 10

__global__ void integrate(double* sum, double step, int threads, int blocks)
{
	// Определяем индекс в линейном массиве по формуле
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	double x = 0;
	for (int i = idx; i < KS; i += threads * blocks)
	{
		x = (i + .5) * step;
		sum[idx] = sum[idx] + 4.0 / (1. + x * x);
	}
}

int main()
{
	int deviceCount = 0;

	printf("Starting...");

	hipError_t error = hipGetDeviceCount(&deviceCount);

	if (error != hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error, hipGetErrorString(error));
		return 1;
	}

	deviceCount == 0 ? printf("There are no available CUDA device(s)\n") : printf("%d CUDA Capable device(s) detected\n", deviceCount);

	/*--------- Simple Kernel ---------*/

	int threads = THREADS, blocks = BLOCKS;
	dim3 block(threads);
	dim3 grid(blocks);


	// Объявляем переменные для хранения суммы на хосте и девайсе
	double* sum_h, * sum_d;
	// Определяем шаг
	double step = 1.0f / KS;
	double pi = 0;

	// Выдялем память для host
	sum_h = (double*)malloc(blocks * threads * sizeof(double));

	// Выдялем память для device
	hipMalloc(&sum_d, blocks * threads * sizeof(double));

	integrate << <grid, block >> > (sum_d, step, threads, blocks);

	// Копирование данных с device на host
	hipMemcpy(sum_h, sum_d, blocks * threads * sizeof(double), hipMemcpyDeviceToHost);

	// Сумма результата на хосте
	for (int i = 0;i < threads * blocks; i++)
	{
		pi += sum_h[i];
	}

	pi *= step;
	printf("Pi == %f", pi);

	free(sum_h);
	hipFree(sum_d);

	system("pause");

	return 0;
}